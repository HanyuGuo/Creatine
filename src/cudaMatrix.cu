#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include "../include/cudaMatrix.cuh"


#define GPU


void cudaMatrix::_init(float *data, int numrows, int numcols){
  devData = NULL;
  numRows = numrows;
  numCols = numcols;
  numElems = numRows * numCols;
  hipError_t err;
  setbestcudaDevice();
  hipSetDevice(gpuid);
  if (data != NULL) {
    // std::cout << "got data!\n";
    if (numElems > 0) {
      hipMalloc((void**)&devData, numElems*sizeof(float));
      hipMemcpy(devData,data,numElems*sizeof(float),hipMemcpyHostToDevice);
      err = hipGetLastError();
      if (err != hipSuccess) {
       std::cout << "Couldn't allocate memory\n";
      }
    }
  }
  else {
    // std::cout << "empty data!\n";
    if (numElems > 0) {
      hipMalloc((void**)&devData, numElems*sizeof(float));
      err = hipGetLastError();
      if (err != hipSuccess) {
       std::cout << "Couldn't allocate memory\n";
      }
    }
  }
}

cudaMatrix::cudaMatrix(int numrows, int numcols){
  _init(NULL,numrows,numcols);
}


cudaMatrix::cudaMatrix(float *data, int numrows, int numcols){
   _init(data,numrows,numcols);
}


cudaMatrix::~cudaMatrix(){
  hipError_t err;
  if (numElems > 0 ) {
      hipFree(devData);
      err = hipGetLastError();

      if (err != hipSuccess) {
        std::cout << "Can't free memory\n";
      }
  }
}

 void cudaMatrix::setbestcudaDevice() {
  int num_dev;
  hipGetDeviceCount(&num_dev);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  long max = props.totalGlobalMem;
  gpuid = 0;
  // std :: cout << "mem " << max << "\n";
  std::vector<hipDeviceProp_t> dev_props;
   for (int i = 1; i < num_dev; i++) {
     hipGetDeviceProperties(&props, i);
     // std :: cout << "mem " << props.totalGlobalMem << "\n";
     if (max < props.totalGlobalMem) {
       max = props.totalGlobalMem;
       gpuid = i;
     }
  }
}

void cudaMatrix::setDeviceData(float *data, int elems) {
  hipError_t err;

  if (elems != numElems) {
    std::cout<< "The size of data must be same! Aborting..\n";
    exit(1);
  }

  hipMemcpy(devData,data,elems*sizeof(float),hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Can't write to device data." << '\n';
  }
}


void cudaMatrix::getDeviceData(float *hdata) {
  hipError_t err;
  // if (hdata == NULL) {
  //    *hdata = new float[numElems];
  // }
  hipMemcpy(hdata,devData, numElems*sizeof(float),hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Can't read device data." << '\n';
  }
}



 void cudaMatrix::cudaAdd(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x,grid_dim_y, 1);
    dim3 block(block_dim_x, block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    // std::cout<<"Launching kernel now...\n";
    MatAddKernel<<<grid, block>>>(adata,bdata,resdata,numRows, numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }
  } else {
    printf("Matrix dims must be same, aborting..\n");
    exit(1);
  }

}


void cudaMatrix::cudaWeightedAdd(const cudaMatrix &b, cudaMatrix &c, float scale) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x,grid_dim_y, 1);
    dim3 block(block_dim_x, block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    // std::cout << "Launching Weighted Add..." << '\n';
    WeightedAddKernel<<< grid, block >>>(adata,bdata,resdata,scale,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }
  } else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}



void cudaMatrix::cudaElemWiseMult(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x, grid_dim_y,1);
    dim3 block(block_dim_x,block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout<<"Doing Elt wise mat mul..."<<"\n";
    EltWiseMatMul<<< grid, block >>> (adata, bdata, resdata,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }

  }else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}


void cudaMatrix::cudaElemWiseDivide(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x, grid_dim_y,1);
    dim3 block(block_dim_x,block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout<<"Doing Elt wise mat divide..."<<"\n";
    EltWiseMatDivide<<< grid, block >>> (adata, bdata, resdata,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }

  }else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}


void cudaMatrix::powgpu(int scale){
  hipError_t err;
  int block_x = 512; // for max threads per block
  int grid_x = (numElems-1)/block_x;
  // int grid_y = (numElems-1)/block_x;
  dim3 grid(grid_x,1,1);
  dim3 block(block_x,1,1);
  powgpu_kernel <<<grid, block >>>(devData,numElems,scale);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch power gpu kernel..\n");
  }

}



void cudaMatrix::expgpu(cudaMatrix &tgt) {
  hipError_t err;
  int block_dim_x = 32;
  int block_dim_y = 32;
  int grid_dim_x = numRows/block_dim_x + 1;
  int grid_dim_y = numCols/block_dim_y + 1;
  // std:: cout << numRows << " " << numCols << "\n";
  dim3 grid(grid_dim_x, grid_dim_y,1);
  dim3 block(block_dim_x,block_dim_y);
  expgpu_kernel <<<grid, block >>>(devData, numElems, tgt.getDevData());
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch exp gpu kernel..\n");
  }
}


void cudaMatrix::axpy_ongpu(const cudaMatrix &b, float scaleA, int ldx, int ldy,cudaMatrix &tgt){
  hipError_t err;
  int block_x = 512;
  int grid_x = (numElems-1)/block_x;
  dim3 grid(grid_x,1,1);
  dim3 block(block_x,1,1);
  printf("Launching axpy kernel...\n");
  axpy_kernel <<< grid, block >>> (devData,b.getDevData(),scaleA,ldx,ldy,numElems,tgt.getDevData());
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch axpy gpu kernel..\n");
  }
}

// float* cudaMatrix::reshape_data(float *data, int numRows, int numCols) {
//     float *rdata;
//
//     hipMalloc((void**)&rdata, numRows*numCols*sizeof(float));
//     for (int i = 0; i < numRows; i++) {
//       for (int j = 0; j < numCols; j++) {
//          rdata[ci(i,j,numCols)] = data[i*numRows+j];
//       }
//     }
//     return rdata;
// }

void cudaMatrix::gemm_ongpu(bool tA, bool tB, const cudaMatrix &b, float scaleA, float scaleB, cudaMatrix &tgt){
  hipError_t err;
  int m = b.getNumCols();
  int n = numRows;
  int k = numCols;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  if ((tgt.getNumCols() != b.getNumCols()) || (tgt.getNumRows()!=numRows)) {
     std::cout << "Matrix dimensions are not same .. aborting \n";
     exit(1);
  }

  hipblasSgemm(handle,(tA?HIPBLAS_OP_T:HIPBLAS_OP_N),(tB?HIPBLAS_OP_T:HIPBLAS_OP_N),
              m,n,k,&scaleA,b.getDevData(),b.getNumCols(),devData,numCols,&scaleB,tgt.getDevData(),b.getNumCols());
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Cannot do Sgemm..\n");
  }

}
void cudaMatrix::cudaAddv(const cudaMatrix &b, float scale, cudaMatrix &c) {
    hipError_t err;
    // int block_dim_x = 4;
    // int block_dim_y = 4;
    // int grid_dim_x = (numRows*numCols)/block_dim_x;
    // int grid_dim_y = (numRows*numCols)/block_dim_y;

    // dim3 grid(grid_dim_x, grid_dim_y,1);
    // dim3 block(block_dim_x,block_dim_y);
    // add_mat_vec_kernel <<<grid, block >>> (devData,b.getDevData(),numRows,numCols,scale,c.getDevData());
    int block_dim_x = 512;
    int grid_dim_x = (numRows*numCols)/block_dim_x + 1;

    dim3 grid(grid_dim_x, 1,1);
    dim3 block(block_dim_x,1);
    add_mat_vec_kernel <<<grid, block >>> (devData,b.getDevData(),numRows,numCols,c.getDevData());
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("can't add matrices and vectors.\n");
    }



}


void cudaMatrix::calc_activation_gpu(Activation a, cudaMatrix &tgt) {
   activations_on_gpu(devData,numElems,a,tgt.getDevData());
}


void cudaMatrix::cudaDivideByVector(const cudaMatrix &b, cudaMatrix &tgt){
     hipError_t err;
     int block_x = 32;
     int block_y = 32;
     int grid_x = (numElems)/block_x;
     dim3 grid(grid_x,1,1);
     dim3 block(block_x,block_y);
     DivideByScalar <<< grid, block >>> (devData,b.getDevData(),numRows, numRows, tgt.getDevData());
     err = hipGetLastError();
     printf("err %d",err);
     if (err != hipSuccess) {
       printf("can't divide by scalar.\n");
     }
}


/* b must be a unit vector for this to work! */
void cudaMatrix::softmax_gpu(cudaMatrix &tgt){
  // hipError_t err1,err2,err3,err4,err5;
  float * temp_max;
  hipMalloc((void**)&temp_max, numRows*sizeof(float));
  float * temp_subtract;
  hipMalloc((void**)&temp_subtract, numRows*numCols*sizeof(float));
  float * temp_exp;
  hipMalloc((void**)&temp_exp, numRows*numCols*sizeof(float));
  // std::cout << "calculating max....";
  int block,grid;
  this->getkernelConfig(true,&block, &grid);
  calc_max <<<grid,block>>>(numRows,numCols,devData,temp_max);
  // err1 = hipGetLastError();
  // std::cout<< "err1 "<< err1;
  this->getkernelConfig(false, &block, &grid);
  subtract_max <<<grid,block>>>(numRows,numCols,devData,temp_max, temp_subtract);
  // err2 = hipGetLastError();
  // std::cout<< "err2 "<< err2;
  this->getkernelConfig(false, &block, &grid);
  expgpu_kernel<<<grid,block>>>(temp_subtract,numRows*numCols, temp_exp);
  // err3 = hipGetLastError();
  // std::cout<<"err3 "<<err3;
  this->getkernelConfig(true, &block, &grid);
  calc_sum_row<<<grid,block>>>(numRows,numCols,temp_exp, temp_max);
  // err4 = hipGetLastError();
  // std::cout<< "err4 "<< err4;
  this->getkernelConfig(false, &block, &grid);
  div_row<<<grid,block>>>(numRows,numCols,temp_exp,temp_max,tgt.getDevData());
  // err5 = hipGetLastError();
  // std::cout<< "err5 "<< err5; 
  // hipFree(scale);
  hipFree(temp_max);
  hipFree(temp_subtract);
  hipFree(temp_exp);

}


void cudaMatrix::argmax_gpu(int* result) {
  int block,grid;
  int * temp_result;
  hipMalloc((void**)&temp_result, numRows*sizeof(int));
  this->getkernelConfig(true, &block, &grid);
  argmax <<<grid,block>>>(numRows,numCols,devData, temp_result);
  hipMemcpy(result,temp_result, numRows*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(temp_result);

}