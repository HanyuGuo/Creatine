#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include "../include/cudaMatrix.cuh"


#define GPU


void cudaMatrix::_init(float *data, int numrows, int numcols){
  numRows = numrows;
  numCols = numcols;
  numElems = numRows * numCols;
  hipError_t err;

  if (devData != NULL) {
    std::cout << "got data!";
  }
  if (numElems > 0 && devData != NULL) {
     hipMalloc((void**)&devData, numElems*sizeof(float));
     hipMemcpy(devData,data,numElems*sizeof(float), hipMemcpyHostToDevice);
     err = hipGetLastError();
     if (err != hipSuccess) {
       std::cout << "Couldn't allocate memory\n";
     }
  }
}

cudaMatrix::cudaMatrix(int numrows, int numcols){
  hipError_t err;
  _init(NULL,numrows,numcols);
  if (numRows*numCols > 0) {
    hipMalloc((void**)&devData, numElems*sizeof(float));
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cout << "Couldn't allocate memory\n";
    }
  }
}


cudaMatrix::cudaMatrix(float *data, int numrows, int numcols){
   _init(data,numrows,numcols);
}


cudaMatrix::~cudaMatrix(){
  hipError_t err;
  if (numElems > 0 ) {
      hipFree(devData);
      err = hipGetLastError();

      if (err != hipSuccess) {
        std::cout << "Can't free memory\n";
      }
  }
}

void cudaMatrix::setDeviceData(float *data, int elems) {
  hipError_t err;
  if (elems != numElems) {
    std::cout<< "The size of data must be same! Aborting..\n";
    exit(1);
  }

  hipMemcpy(devData,data,elems*sizeof(float),hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Can't write to device data." << '\n';
  }
}


void cudaMatrix::getDeviceData(float *hdata) {
  hipError_t err;
  // if (hdata == NULL) {
  //    *hdata = new float[numElems];
  // }
  hipMemcpy(hdata,devData, numElems*sizeof(float),hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Can't read device data." << '\n';
  }
}



void cudaMatrix::cudaAdd(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x,grid_dim_y, 1);
    dim3 block(block_dim_x, block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout<<"Launching kernel now...\n";
    MatAddKernel<<<grid, block>>>(adata,bdata,resdata,numRows, numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }
  } else {
    printf("Matrix dims must be same, aborting..\n");
    exit(1);
  }

}


void cudaMatrix::cudaWeightedAdd(const cudaMatrix &b, cudaMatrix &c, float scale) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x,grid_dim_y, 1);
    dim3 block(block_dim_x, block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout << "Launching Weighted Add..." << '\n';
    WeightedAddKernel<<< grid, block >>>(adata,bdata,resdata,scale,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }
  } else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}



void cudaMatrix::cudaElemWiseMult(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x, grid_dim_y,1);
    dim3 block(block_dim_x,block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout<<"Doing Elt wise mat mul..."<<"\n";
    EltWiseMatMul<<< grid, block >>> (adata, bdata, resdata,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }

  }else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}


void cudaMatrix::cudaElemWiseDivide(const cudaMatrix &b, cudaMatrix &c) {
  hipError_t err;
  if (numRows == b.getNumRows() && numCols == b.getNumCols()) {
    int block_dim_x = 32;
    int block_dim_y = 32;
    int grid_dim_x = (numRows*numCols)/block_dim_x;
    int grid_dim_y = (numRows*numCols)/block_dim_y;
    dim3 grid(grid_dim_x, grid_dim_y,1);
    dim3 block(block_dim_x,block_dim_y);
    float *adata = devData;
    float *bdata = b.getDevData();
    float *resdata = c.getDevData();
    std::cout<<"Doing Elt wise mat divide..."<<"\n";
    EltWiseMatDivide<<< grid, block >>> (adata, bdata, resdata,numRows,numCols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error launching kernel\n");
    }

  }else {
    printf("Matrix dims must be the same, aborting...\n");
    exit(1);
  }
}


void cudaMatrix::powgpu(int scale, int n){
  hipError_t err;
  int block_x = 512; // for max threads per block
  int grid_x = (numElems-1)/block_x;
  dim3 grid(grid_x,1,1);
  dim3 block(block_x,1,1);
  powgpu_kernel <<<grid, block >>>(devData,n,scale);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch power gpu kernel..\n");
  }

}



void cudaMatrix::expgpu(int n) {
  hipError_t err;
  int block_x = 512; // for max threads per block
  int grid_x = (numElems-1)/block_x;
  dim3 grid(grid_x,1,1);
  dim3 block(block_x,1,1);
  expgpu_kernel <<<grid, block >>>(devData, n);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch exp gpu kernel..\n");
  }
}


void cudaMatrix::axpy_ongpu(const cudaMatrix &b, float scaleA, int ldx, int ldy,cudaMatrix &tgt){
  hipError_t err;
  int block_x = 512;
  int grid_x = (numElems-1)/block_x;
  dim3 grid(grid_x,1,1);
  dim3 block(block_x,1,1);
  printf("Launching axpy kernel...\n");
  axpy_kernel <<< grid, block >>> (devData,b.getDevData(),scaleA,ldx,ldy,numElems,tgt.getDevData());
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Couldn't launch axpy gpu kernel..\n");
  }
}

void cudaMatrix::gemm_ongpu(bool tA, bool tB, const cudaMatrix &b, float scaleA, float scaleB, cudaMatrix &tgt) const{
  hipError_t err;
  int m = b.getNumCols();
  int k = numCols;
  int n = numRows;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

    hipblasSgemm(handle,(tA?HIPBLAS_OP_T:HIPBLAS_OP_N),(tB?HIPBLAS_OP_T:HIPBLAS_OP_N),
                m,n,k,&scaleA,b.getDevData(),b.getLeadingDim(),devData,numCols,&scaleB,tgt.getDevData(),tgt.getLeadingDim());
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Cannot do Sgemm..\n");
    }


}
