#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "../include/cudaKernels.cuh"


__global__ void MatAddKernel(float *a, float *b, float *c, int numRows,int numCols){
int idx = blockIdx.x*blockDim.x+threadIdx.x;
int idy = blockIdx.y*blockDim.y+threadIdx.y;
if(idy < numRows && idx < numCols) {
  c[idy*numRows+idx] = a[idy*numRows+idx]+b[idy*numRows+idx];
}
}

__global__ void WeightedAddKernel(float *a, float *b, float *c,float scale,int numRows,int numCols){
int idx = blockIdx.x*blockDim.x+threadIdx.x;
int idy = blockIdx.y*blockDim.y+threadIdx.y;
if(idy < numRows && idx < numCols) {
  c[idy*numRows+idx] = a[idy*numRows+idx]+ scale*b[idy*numRows+idx];
}
}

__global__ void EltWiseMatMul(float *a, float *b, float *c,int numRows, int numCols) {
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int idy = blockIdx.y*blockDim.y+threadIdx.y;
  if(idy < numRows && idx < numCols) {
    c[idy*numRows+idx] = a[idy*numRows+idx]*b[idy*numRows+idx];
  }
}


__global__ void EltWiseMatDivide(float *a, float *b, float *c,int numRows, int numCols) {
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int idy = blockIdx.y*blockDim.y+threadIdx.y;
  if(idy < numRows && idx < numCols) {
    c[idy*numRows+idx] = a[idy*numRows+idx]/b[idy*numRows+idx];
  }
}


__global__ void  powgpu_kernel(float *a, int n,  int scale){
  int id = (blockIdx.x + gridDim.x*blockIdx.y)*blockDim.x + threadIdx.x;
  if (id < n) {
    a[id] = pow(a[id],scale);
  }
}

__global__ void expgpu_kernel(float *a, int n){
  int id = (blockIdx.x + gridDim.x*blockIdx.y)*blockDim.x + threadIdx.x;
  if (id < n) {
    a[id] = exp(a[id]);
  }
}


__global__ void axpy_kernel(float *a, float *b, float scaleA,int lda,int ldy, int n, float *y){
  int id = (blockIdx.x + gridDim.x*blockIdx.y)*blockDim.x + threadIdx.x;
  if (id<n) {
       y[id*ldy] = scaleA*a[id*lda]+b[id*lda];
      //  printf("%.2f ",y[id*ldy]);
  }
}

__global__ void add_mat_vec_kernel(float *a, float *b, int nr, int nc,float scale, float *y){
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    if (idx < nc && idy < nr) {
        y[idy*nr+idx] = a[idy*nr+idx]+b[idx];
    }
}


__global__ void softmax_kernel(float *x, int nr, int nc){
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int idy = blockIdx.y*blockDim.y+threadIdx.y;
  float sum = 0;
  if (idx < nc && idy < nr) {
     sum += x[idx*nr+idy]; // sum of all cols.
  }
  x[idy*nr+idy] /= sum;
}
