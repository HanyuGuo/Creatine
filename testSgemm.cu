#include<hip/hip_runtime.h>
#include<hipblas.h>
#include <stdio.h>
int ci(int row, int column, int nColumns) {
  return row*nColumns+column;
}
int main(int argc, char const *argv[]) {
  int rowA, rowB, colA, colB,rowC, colC;
  rowA = 5;
  colA = 2;
  rowB = colA;
  colB = 3;
  rowC = rowA;
  colC = colB;
  float *ddata1, *ddata2, *resdata;
  float *data1 = new float[rowA*colA];
  float *data2 = new float[rowB*colB];
  float *res = new float[rowC*colC];

  for (int i = 0; i < rowA; ++i) {
    for (int j = 0; j < colA; j++) {
         data1[ci(i,j,colA)] = i;

    }
}

for (int i = 0; i < rowB; ++i) {
  for (int j = 0; j < colB; j++) {
       data2[ci(i,j,colB)] = i;

  }
}
hipMalloc((void**)&ddata1,rowA*colA*sizeof(float));
hipMalloc((void**)&ddata2,rowB*colB*sizeof(float));
hipMalloc((void**)&resdata,rowC*colC*sizeof(float));
hipMemcpy(ddata1, data1, rowA*colA*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(ddata2, data2, rowB*colB*sizeof(float),hipMemcpyHostToDevice);
hipblasHandle_t handle;
hipblasCreate(&handle);
float alpha = 1.0f;
float beta = 0.0f;
hipError_t err;
hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,colB,rowA,colA,&alpha,ddata2,colB,ddata1,colA,&beta,resdata,colB);
err = hipGetLastError();
if (err != hipSuccess) {
  printf("Can't perform Sgemm...\n");
}
hipMemcpy(res, resdata, rowC*colC*sizeof(float),hipMemcpyDeviceToHost);
for (int i = 0; i < rowC; i++) {
  for (int j = 0; j < colC; j++) {
       printf("%.2f ",res[ci(i,j,colC)]);

  }
   printf("\n");
}

  return 0;
}
